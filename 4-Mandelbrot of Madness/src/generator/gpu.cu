#include "hip/hip_runtime.h"
#include "../headers/generator/gpu.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

using ComplexCU = float2;    // cuda datatype

// __device__ untuk fungsi yang dipanggil di dalam kernel

__device__ Color getColor_device(int n, int max_iters) {
    const unsigned char bg_r = 64;
    const unsigned char bg_g = 0;
    const unsigned char bg_b = 0;
    
    if (n == max_iters) return {0, 0, 0};

    // normalisasi
    float t = static_cast<float>(n)/ static_cast<float> (max_iters);

    // interpolation
    float interp_r = static_cast<unsigned char>(9 * (1 - t) * t * t * t * 255);
    float interp_g = static_cast<unsigned char>(15 * (1 - t) * (1 - t) * t * t * 255);
    float interp_b = static_cast<unsigned char>(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);

    unsigned char r = static_cast<unsigned char>(min(255.0f, bg_r + interp_r));
    unsigned char g = static_cast<unsigned char>(min(255.0f, bg_g + interp_g));
    unsigned char b = static_cast<unsigned char>(min(255.0f, bg_b + interp_b));

    return {r, g, b};
}

__device__ int calculateMandelbrot_device(const ComplexCU& c, int max_iters) {
    ComplexCU z = {0.0, 0.0};
    int n = 0;
    while (z.x * z.x + z.y * z.y <= 4.0 && n < max_iters) {
        float real_temp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0 * z.x * z.y + c.y;
        z.x = real_temp;
        n++;
    }
    return n;
}

// ==============================          KERNEL            ===============================
__global__ void mandelbrotKernel(Color* pixels, int width, int height, int max_iterations, Bounds bounds) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_pixels = width * height;

    if (index >= num_pixels) {
        return;
    }

    // Turn idx to pixel coords.
    int x = index % width;
    int y = index / width;

    double x_range = bounds.x_max - bounds.x_min;
    double y_range = bounds.y_max - bounds.y_min;

    float real = bounds.x_min + (static_cast<float>(x) / (width - 1)) * x_range;
    float imag = bounds.y_min + (static_cast<float>(y) / (height - 1)) * y_range;
    ComplexCU c = {real, imag};

    int iterations;
    
    if ((real + 1.0) * (real + 1.0) + imag * imag < 0.0625) { // 1/16
        iterations = max_iterations;
    }
    else {
        float q = (real - 0.25) * (real - 0.25) + imag * imag;
        if (q * (q + (real - 0.25)) < 0.25 * imag * imag) {
            iterations = max_iterations;
        } else {
            iterations = calculateMandelbrot_device(c, max_iterations);
        }
    }
    pixels[index] = getColor_device(iterations, max_iterations);
}

Colors mandelbrotGPU(int width, int height, int max_iterations, const Bounds& bounds) {
    int num_pixels = width * height;
    size_t buffer_size = num_pixels * sizeof(Color);

    Color* pixels = nullptr;
    hipMalloc(&pixels, buffer_size);

    int threads = 256;
    int blocks = (num_pixels + threads - 1) / threads;
    mandelbrotKernel<<<blocks, threads>>>(pixels, width, height, max_iterations, bounds);
    
    hipGetLastError(); 

    // Copy device to host (data)
    Colors h_pixels(num_pixels);
    hipMemcpy(h_pixels.data(), pixels, buffer_size, hipMemcpyDeviceToHost);
    hipFree(pixels);

    return h_pixels;
}

__device__ int calculateJulia_device(ComplexCU z, const ComplexCU& c, int max_iters) {
    int n = 0;
    while (z.x * z.x + z.y * z.y <= 4.0f && n < max_iters) {
        float real_temp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0f * z.x * z.y + c.y;
        z.x = real_temp;
        n++;
    }
    return n;
}

__global__ void juliaKernel(Color* pixels, int width, int height, int max_iterations, ComplexCU c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width * height) return;

    int x = index % width;
    int y = index / width;

    // Memetakan koordinat piksel ke bidang kompleks untuk Julia set
    // Rentang [-2, 2] biasanya memberikan hasil yang baik
    float real = -2.0f + (static_cast<float>(x) / (width - 1)) * 4.0f;
    float imag = -2.0f + (static_cast<float>(y) / (height - 1)) * 4.0f;
    ComplexCU z = {real, imag};

    int iterations = calculateJulia_device(z, c, max_iterations);
    pixels[index] = getColor_device(iterations, max_iterations);
}

Colors juliaGPU(int width, int height, int max_iterations, const Complex& c) {
    int num_pixels = width * height;
    size_t buffer_size = num_pixels * sizeof(Color);

    Color* d_pixels = nullptr;
    hipMalloc(&d_pixels, buffer_size);

    // Konversi konstanta Julia dari tipe host (double) ke tipe device (float)
    ComplexCU c_device = {static_cast<float>(c.real()), static_cast<float>(c.imag())};

    int threads = 256;
    int blocks = (num_pixels + threads - 1) / threads;
    juliaKernel<<<blocks, threads>>>(d_pixels, width, height, max_iterations, c_device);

    hipGetLastError(); 

    Colors h_pixels(num_pixels);
    hipMemcpy(h_pixels.data(), d_pixels, buffer_size, hipMemcpyDeviceToHost);
    hipFree(d_pixels);

    return h_pixels;
}
