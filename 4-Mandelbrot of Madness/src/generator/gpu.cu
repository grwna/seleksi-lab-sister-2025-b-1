#include "hip/hip_runtime.h"
#include "../headers/generator/gpu.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

using ComplexCU = double2;    // cuda datatype

// __device__ untuk fungsi yang dipanggil di dalam kernel

__device__ Color getColor_device(int n, int max_iters) {
    if (n == max_iters) return {0, 0, 0};

    // normalisasi
    double t = static_cast<double>(n)/ static_cast<double> (max_iters);

    // interpolation
    unsigned char r = static_cast<unsigned char>(9 * (1 - t) * t * t * t * 255);
    unsigned char g = static_cast<unsigned char>(15 * (1 - t) * (1 - t) * t * t * 255);
    unsigned char b = static_cast<unsigned char>(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);

    return {r, g, b};
}

__device__ int calculateMandelbrot_device(const ComplexCU& c, int max_iters) {
    ComplexCU z = {0.0, 0.0};
    int n = 0;
    while (z.x * z.x + z.y * z.y <= 4.0 && n < max_iters) {
        double real_temp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0 * z.x * z.y + c.y;
        z.x = real_temp;
        n++;
    }
    return n;
}

// ==============================          KERNEL            ===============================
__global__ void mandelbrotKernel(Color* pixels, int width, int height, int max_iterations) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_pixels = width * height;

    if (index >= num_pixels) {
        return;
    }

    // Turn idx to pixel coords.
    int x = index % width;
    int y = index / width;

    double real = -2.0 + (static_cast<double>(x) / (width - 1)) * 3.0;
    double imag = -1.0 + (static_cast<double>(y) / (height - 1)) * 2.0;
    ComplexCU c = {real, imag};

    int iterations;
    
    if ((real + 1.0) * (real + 1.0) + imag * imag < 0.0625) { // 1/16
        iterations = max_iterations;
    }
    else {
        double q = (real - 0.25) * (real - 0.25) + imag * imag;
        if (q * (q + (real - 0.25)) < 0.25 * imag * imag) {
            iterations = max_iterations;
        } else {
            iterations = calculateMandelbrot_device(c, max_iterations);
        }
    }
    pixels[index] = getColor_device(iterations, max_iterations);
}

Colors mandelbrotGPU(int width, int height, int max_iterations) {
    int num_pixels = width * height;
    size_t buffer_size = num_pixels * sizeof(Color);

    Color* pixels = nullptr;
    hipMalloc(&pixels, buffer_size);

    int threads = 256;
    int blocks = (num_pixels + threads - 1) / threads;
    mandelbrotKernel<<<blocks, threads>>>(pixels, width, height, max_iterations);
    
    hipGetLastError(); 

    // Copy device to host (data)
    Colors h_pixels(num_pixels);
    hipMemcpy(h_pixels.data(), pixels, buffer_size, hipMemcpyDeviceToHost);
    hipFree(pixels);

    return h_pixels;
}