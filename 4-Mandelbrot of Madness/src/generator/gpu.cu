#include "hip/hip_runtime.h"
#include "../headers/generator/gpu.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

using ComplexCU = float2;    // cuda datatype

// __device__ untuk fungsi yang dipanggil di dalam kernel

__device__ Color getColor_device(int n, int max_iters, ColorScheme scheme) {
    if (n == max_iters) return {0, 0, 0};

    float t = static_cast<float>(n) / static_cast<float>(max_iters);

    float r_coeff1 = scheme.pattern_r[0];
    float r_coeff2 = scheme.pattern_r[1];
    float r_coeff3 = scheme.pattern_r[2];
    
    float g_coeff1 = scheme.pattern_g[0];
    float g_coeff2 = scheme.pattern_g[1];
    float g_coeff3 = scheme.pattern_g[2];

    float b_coeff1 = scheme.pattern_b[0];
    float b_coeff2 = scheme.pattern_b[1];
    float b_coeff3 = scheme.pattern_b[2];

    float interp_r = static_cast<unsigned char>(r_coeff1 * pow(1 - t, r_coeff2) * pow(t, r_coeff3) * 255);
    float interp_g = static_cast<unsigned char>(g_coeff1 * pow(1 - t, g_coeff2) * pow(t, g_coeff3) * 255);
    float interp_b = static_cast<unsigned char>(b_coeff1 * pow(1 - t, b_coeff2) * pow(t, b_coeff3) * 255);

    unsigned char r = static_cast<unsigned char>(::min(255.0f, (scheme.background[0] * 255) + interp_r));
    unsigned char g = static_cast<unsigned char>(::min(255.0f, (scheme.background[1] * 255) + interp_g));
    unsigned char b = static_cast<unsigned char>(::min(255.0f, (scheme.background[2] * 255) + interp_b));

    return {r, g, b};
}

__device__ int calculateMandelbrot_device(const ComplexCU& c, int max_iters) {
    ComplexCU z = {0.0, 0.0};
    int n = 0;
    while (z.x * z.x + z.y * z.y <= 4.0 && n < max_iters) {
        float real_temp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0 * z.x * z.y + c.y;
        z.x = real_temp;
        n++;
    }
    return n;
}

// ==============================          KERNEL            ===============================
__global__ void mandelbrotKernel(Color* pixels, int width, int height, int max_iterations, Bounds bounds, ColorScheme scheme) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_pixels = width * height;

    if (index >= num_pixels) {
        return;
    }

    // Turn idx to pixel coords.
    int x = index % width;
    int y = index / width;

    double x_range = bounds.x_max - bounds.x_min;
    double y_range = bounds.y_max - bounds.y_min;

    float real = bounds.x_min + (static_cast<float>(x) / (width - 1)) * x_range;
    float imag = bounds.y_min + (static_cast<float>(y) / (height - 1)) * y_range;
    ComplexCU c = {real, imag};

    int iterations;
    
    if ((real + 1.0) * (real + 1.0) + imag * imag < 0.0625) { // 1/16
        iterations = max_iterations;
    }
    else {
        float q = (real - 0.25) * (real - 0.25) + imag * imag;
        if (q * (q + (real - 0.25)) < 0.25 * imag * imag) {
            iterations = max_iterations;
        } else {
            iterations = calculateMandelbrot_device(c, max_iterations);
        }
    }
    pixels[index] = getColor_device(iterations, max_iterations, scheme);
}

Colors mandelbrotGPU(int width, int height, int max_iterations, const Bounds& bounds, const ColorScheme& scheme) {
    int num_pixels = width * height;
    size_t buffer_size = num_pixels * sizeof(Color);

    Color* pixels = nullptr;
    hipMalloc(&pixels, buffer_size);

    int threads = 256;
    int blocks = (num_pixels + threads - 1) / threads;
    mandelbrotKernel<<<blocks, threads>>>(pixels, width, height, max_iterations, bounds, scheme);
    
    hipGetLastError(); 

    // Copy device to host (data)
    Colors h_pixels(num_pixels);
    hipMemcpy(h_pixels.data(), pixels, buffer_size, hipMemcpyDeviceToHost);
    hipFree(pixels);

    return h_pixels;
}

__device__ int calculateJulia_device(ComplexCU z, const ComplexCU& c, int max_iters) {
    int n = 0;
    while (z.x * z.x + z.y * z.y <= 4.0f && n < max_iters) {
        float real_temp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0f * z.x * z.y + c.y;
        z.x = real_temp;
        n++;
    }
    return n;
}

__global__ void juliaKernel(Color* pixels, int width, int height, int max_iterations, ComplexCU c, ColorScheme scheme) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= width * height) return;

    int x = index % width;
    int y = index / width;

    // Memetakan koordinat piksel ke bidang kompleks untuk Julia set
    // Rentang [-2, 2] biasanya memberikan hasil yang baik
    float real = -2.0f + (static_cast<float>(x) / (width - 1)) * 4.0f;
    float imag = -2.0f + (static_cast<float>(y) / (height - 1)) * 4.0f;
    ComplexCU z = {real, imag};

    int iterations = calculateJulia_device(z, c, max_iterations);
    pixels[index] = getColor_device(iterations, max_iterations, scheme);
}

Colors juliaGPU(int width, int height, int max_iterations, const Complex& c, const ColorScheme& scheme) {
    int num_pixels = width * height;
    size_t buffer_size = num_pixels * sizeof(Color);

    Color* d_pixels = nullptr;
    hipMalloc(&d_pixels, buffer_size);

    // Konversi konstanta Julia dari tipe host (double) ke tipe device (float)
    ComplexCU c_device = {static_cast<float>(c.real()), static_cast<float>(c.imag())};

    int threads = 256;
    int blocks = (num_pixels + threads - 1) / threads;
    juliaKernel<<<blocks, threads>>>(d_pixels, width, height, max_iterations, c_device, scheme);

    hipGetLastError(); 

    Colors h_pixels(num_pixels);
    hipMemcpy(h_pixels.data(), d_pixels, buffer_size, hipMemcpyDeviceToHost);
    hipFree(d_pixels);

    return h_pixels;
}
